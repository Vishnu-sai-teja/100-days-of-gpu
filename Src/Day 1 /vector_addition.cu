
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vector_addition_kernel(float *A, float *B, float *C, int n){
  int i = blockDim.x * blockIdx.x + threadIdx.x ;
  if (i < n){
    C[i] = A[i] + B[i]; 
  }
}

__host__
void vector_addition_host(float *A, float *B, float *C, int n){
  // Create the varaiables inside the device and copy the values from host to device
  float *A_d, *B_d, *C_d;
  int size = n * sizeof(float);
  hipMalloc((void **) &A_d, size);
  hipMalloc((void **) &B_d, size);
  hipMalloc((void **) &C_d, size);

  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

  // process vector vector_addition
  vector_addition_kernel <<<int(ceil(n/256.0)), 256>>>(A_d, B_d, C_d, n);

  // copy back from device to host and free the memory
  hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  for(int i=0; i<5; i++){
    printf("C at %d position is %f\n", i, C[i]);
  }
}
int main(){
  float *A, *B, *C;

  float X[5] = {1, 2, 3, 4, 5};
  float Y[5] = {6, 7, 8, 9, 10};
  float Z[5] = {};

  A = &X[0];
  B = &Y[0];
  C = &Z[0];
  vector_addition_host(A, B, C, 5);
}